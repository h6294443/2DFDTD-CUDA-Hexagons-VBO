#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "parameters.h"
#include "grid-2d.h"
#include <stdio.h>



//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t addWithCuda(Grid *g);

__global__ void addKernel(double *c, double *a, double *b)
{
	int i = threadIdx.x;
	int offset = 1;
	double temp = c[i];
	if (i == 9) 
		offset = 0;
	c[i] = a[i] + b[i] + temp;
	__syncthreads();

}

int main()
{
	hipError_t cudaStatus;
	Grid *g = new Grid;
	gridInit(g);
	const int arraySize = g->sizeX*(g->sizeY-1);
	

for (int i = 0; i < 10; i++) {
	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(g);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "addWithCuda failed!");
	return 1;
	}
	for (int j = 0; j<arraySize-1;j++) 
		printf("%g + %g = %g\n", g->chxh[j], g->chxe[j], g->hx[j]);
	//printf("test\n");
		
}
	
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
hipError_t addWithCuda(Grid *g)
{
    double *dev_chxh = 0;
    double *dev_chxe = 0;
    double *dev_hx = 0;
	hipError_t cudaStatus;
	int size = g->sizeX * (g->sizeY - 1);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_hx, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	
	cudaStatus = hipMalloc((void**)&dev_chxh, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_chxe, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_chxh, g->chxh, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_hx, g->hx, size * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_chxe, g->chxe, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_hx, dev_chxh, dev_chxe);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(g->hx, dev_hx, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_hx);
	hipFree(dev_chxh);
    hipFree(dev_chxe);
    
    return cudaStatus;
}
